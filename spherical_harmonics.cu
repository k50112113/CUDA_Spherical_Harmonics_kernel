#include "hip/hip_runtime.h"
// Reference
// https://github.com/NVIDIA/torch-harmonics
// Schaeffer, N.; Efficient spherical harmonic transforms aimed at pseudospectral numerical simulations, G3: Geochemistry, Geophysics, Geosystems.

#include <cmath>
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
const uint max_allowed_n = 13;

__host__
void compute_associated_legendre_polynomial_coefficients(
  uint n,
  vector<int>& a_n_m,
  vector<int>& b_n_m)
{
  // precompute the coefficients for associated legendre polynomials on host
  a_n_m.resize((n + 1) * (n + 1), 0.0);
  b_n_m.resize((n + 1) * (n + 1), 0.0);
  a_n_m[0] = 1.0;
  for (uint l = 1; l <= n; ++l) {
    for (uint m = 0; m < l-1; ++m) {
      a_n_m[l * (n + 1) + m] = sqrt((2.0*l - 1) / (l - m) * (2.0*l + 1) / (l + m));
      b_n_m[l * (n + 1) + m] = sqrt(((float)l + m - 1) / (l - m) * (2.0*l + 1) / (2.0*l - 3) * (l - m - 1) / (l + m));
    }
    a_n_m[l * (n + 1) + l - 1] = sqrt(2.0*l + 1);
    a_n_m[l * (n + 1) + l]     = sqrt((2.0*l + 1) / 2.0 / l);
    // the following is the true value for a_n_m(l, l) but the cumulative products are done during the generation process of the polynomials
    // a_n_m[l * (n + 1) + l]  = sqrt((2.0*l + 1) / 2.0 / l) * a_n_m[(l - 1) * (n + 1) + l - 1];
  }
}

__device__
void compute_associated_legendre_polynomial_kernel(
  uint n,
  const float* __restrict__ a_n_m,
  const float* __restrict__ b_n_m,
  float* p_m,
  float x)
{
  // return Pnm at selected n from m = 0 ~ n on device
  float p_m_last_three_rows[3*max_allowed_n];
  p_m_last_three_rows[0] = 1.0;
  for (uint l = 1; l <= n; ++l) {
    uint l0 = l % 3;
    uint l1 = (l-1) % 3;
    uint l2 = (l-2) % 3;
    for (uint m = 0; m < l-1; ++m) {
      p_m_last_three_rows[l0 * max_allowed_n + m] = x * a_n_m[l * (n + 1) + m] * p_m_last_three_rows[l1 * max_allowed_n + m] -
                                                                 b_n_m[l * (n + 1) + m] * p_m_last_three_rows[l2 * max_allowed_n + m];
    }
    p_m_last_three_rows[l0 * max_allowed_n + l-1] =                           x * a_n_m[l * (n + 1) + l-1] * p_m_last_three_rows[l1 * max_allowed_n + l-1];                     
    p_m_last_three_rows[l0 * max_allowed_n + l  ] = sqrt((1.0 + x) * (1.0 - x)) * a_n_m[l * (n + 1) + l  ] * p_m_last_three_rows[l1 * max_allowed_n + l-1];
  }
  for (uint m = 0; m <= n; ++m) {
    p_m[m] = p_m_last_three_rows[(n % 3) * max_allowed_n + m];
  }
  if (n + 1 >= 2) {
    for (uint m = 0; m <= n; ++m) {
      p_m[m] /= a_n_m[n * (n + 1) + n-1];
    }
  }
  for (uint m = 1; m <= n; m+=2){
    p_m[m] *= -1.0;
  }
}

__device__
void compute_spherical_harmonics_kernel(
  uint n,
  const float* __restrict__ a_n_m,
  const float* __restrict__ b_n_m,
  float theta,
  float phi,
  float* Y_m_real,
  float* Y_m_imag)
{
  // return Ynm at selected n from m = 0 ~ n on device
  // theta [0, pi]
  // phi   [0, 2pi)
  float p_m[max_allowed_n];
  compute_associated_legendre_polynomial_kernel(n, a_n_m, b_n_m, p_m, cos(theta));
  float pre_factor = sqrt((2.0*n + 1) / (4.0 * M_PI));
  for (uint i_m = 0; i_m <= n; ++i_m) {
    atomicAdd(&Y_m_real[i_m], pre_factor * p_m[i_m] * cos(i_m * phi));
    atomicAdd(&Y_m_imag[i_m], pre_factor * p_m[i_m] * sin(i_m * phi));
  }
}
